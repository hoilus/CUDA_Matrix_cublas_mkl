#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cstdio>

using namespace std;

// Thread block size
const int blockSize = 16;

// Matrices are stored in row-major order:
// M(row, clo) = *(M.elements + row*M.width + col);
typedef struct {
  int width;
  int height;
  float* elements;
} Matrix;

// CPU matrix multiplication for evaluating results
void cpu_matrix_multi(float *matA, float *matB, float *matC, int m, int k, int n) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      float tmp = 0.0;
      for (int l = 0; l < k; l++) {
        tmp += matA[i*k + l] * matB[l*n + j];
      }
      matC[i*n + j] = tmp;
    }
  }
}

// Matrix multiplication kernel called by MatMul()
__global__
void MatMulKernel_naive(const Matrix A, const Matrix B, Matrix C) {
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  float Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  for (int e = 0; e < A.width; e++)
    Cvalue += A.elements[row*A.width+e] * B.elements[e*B.width+col];
  C.elements[row*C.width+col] = Cvalue;
}

// Matrix multiplication - host code
// Matrix dimensions are assumed to be multiples of blockSize
void MatMul(const Matrix A, const Matrix B, Matrix C) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width; d_B.height = B.width;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width; d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);
  
  // Invoke kernel
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y, 1);
  MatMulKernel_naive<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

int main() {
  // Initiate A and B elements on host memory
  Matrix h_A;
  h_A.height = 1024; h_A.width = 1024;
  float* h_matA = new float[h_A.height * h_A.width];
  std::srand(1103);
  for (int i = 0; i < h_A.height; i++)
    for (int j = 0; j < h_A.width; j++)
      h_matA[i*h_A.width+j] = float(std::rand())/float(RAND_MAX);
  h_A.elements = h_matA;

  Matrix h_B;
  h_B.height = 1024; h_B.width = 1024;
  float* h_matB = new float[h_B.height * h_B.width];
  for (int i = 0; i < h_B.height; i++)
    for (int j = 0; j < h_B.width; j++)
      h_matB[i*h_B.width+j] = float(std::rand())/float(RAND_MAX);
  h_B.elements = h_matB;

  // Matrix C size
  Matrix h_C;
  h_C.height = h_A.height; h_C.width = h_B.width;
  float* h_matC = new float[h_A.height * h_B.width];
  h_C.elements = h_matC;

  // Call MatMul()
  MatMul(h_A, h_B, h_C);

  // Evaluate results
  float* h_matC_cpu = new float[h_A.height * h_B.width];
//  cpu_matrix_multi(h_matA, h_matB, h_matC_cpu, h_A.height, h_A.width, h_B.width);
  cpu_matrix_multi(h_A.elements, h_B.elements, h_matC_cpu, h_A.height, h_A.width, h_B.width);
  bool res_flag = false;
  float resol = 0.000001;
  for (int i = 0; i < h_C.height; i++) {
    for (int j = 0; j < h_C.width; j++) {
      if (fabs(*(h_C.elements+i*h_C.width+j) - h_matC[i*h_C.width+j]) > resol)
	res_flag = true;
    }
  }
	
  if (res_flag == false)
    cout << "Matrix multiplication by GPU is right! " << endl;
  else
    cout << "Results are not right! " << endl;

  // Free memory on host 
  delete [] h_matA;
  delete [] h_matB;
  delete [] h_matC;
  delete [] h_matC_cpu;

  return 0;
}

